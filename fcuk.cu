#include "hip/hip_runtime.h"
#include "fcuk.h"

__device__ inline bool islower(char c) { return c >= 'a' && c <= 'z'; }

__device__ inline bool isupper(char c) { return c >= 'A' && c <= 'Z'; }

__device__ inline bool isdigit(char c) { return c >= '0' && c <= '9'; }

__device__ inline bool isspecial(char c) {
  return c == '/' || c == '-' || c == '_' || c == ' ' || c == '.';
}

__device__ inline char tolower(char c) {
  return isupper(c) ? c - 'A' + 'a' : c;
}

bool init = false;
__constant__ score_t SPECIAL_BONUS_C[256];

__global__ void match_kernel(char *__restrict__ buf,
                             size_t *__restrict__ indices, size_t buf_size,
                             size_t n_sources, const char *__restrict__ pattern,
                             size_t n_pattern, bool *has_match) {
  size_t idx = indices[blockIdx.x];
  string_t source = {
      .data = buf + idx,
      .len =
          (blockIdx.x == n_sources - 1 ? buf_size : indices[blockIdx.x + 1]) -
          idx};
  int32_t j = threadIdx.x;

  if (j >= source.len)
    return;

  // // sequential
  // if (j == 0) {
  //   for (size_t i = 0; i < source.len; ++i) {
  //     if (tolower(source.data[i]) == *pattern)
  //       ++pattern;
  //   }
  //   has_match[blockIdx.x] = *pattern == '\0';
  // }

  if (j == 0) {
    has_match[blockIdx.x] = true;
  }

  __shared__ size_t pos[MAX_STR_LEN]; // TODO: wasting resource, change to n_str
  int32_t prev = -1;

  for (size_t i = 0; i < n_pattern; ++i) {
    pos[j] = tolower(source.data[j]) == tolower(pattern[i]) && j > prev
                 ? j
                 : INT32_MAX;

    __syncthreads();

    for (size_t n = source.len, s = (n + 1) / 2; s > 0;
         n = s, s = (s + 1) / 2) {
      if (j < s && j + s < n) {
        pos[j] = min(pos[j], pos[j + s]);
      }
      __syncthreads();
      if (s == 1)
        break;
    }

    prev = pos[0];

    if (prev == INT32_MAX) {
      if (j == 0) {
        has_match[blockIdx.x] = false;
      }
      return;
    }

    __syncthreads();
  }
}

__global__ void fused_score_kernel(char *__restrict__ buf,
                                   size_t *__restrict__ indices,
                                   size_t buf_size, size_t n_sources,
                                   const char *__restrict__ pattern,
                                   size_t n_pattern, score_t *res_scores) {
  size_t idx = indices[blockIdx.x];
  string_t source = {
      .data = buf + idx,
      .len =
          (blockIdx.x == n_sources - 1 ? buf_size : indices[blockIdx.x + 1]) -
          idx};
  size_t j = threadIdx.x;

  if (j >= source.len)
    return;

  if (source.len > 1024 || n_pattern > 1024) {
    // strings too long
    if (j == 0) {
      res_scores[blockIdx.x] = SCORE_MIN;
    }
    return;
  }

  if (source.len == n_pattern) {
    // this function is only called when str contains the
    // pattern
    if (j == 0) {
      res_scores[blockIdx.x] = SCORE_MAX;
    }
    return;
  }

  __shared__ score_t
      match_bonus_s[MAX_STR_LEN]; // TODO: wasting resource, change to n_str
  __shared__ score_t M_s[3][MAX_STR_LEN], D_s[3][MAX_STR_LEN];

  M_s[0][j] = SCORE_MIN;
  M_s[1][j] = SCORE_MIN;
  M_s[2][j] = SCORE_MIN;

  D_s[0][j] = SCORE_MIN;
  D_s[1][j] = SCORE_MIN;
  D_s[2][j] = SCORE_MIN;

  if (j < source.len) {
    char curr = source.data[j], prev = j > 0 ? source.data[j - 1] : '/';

    if (islower(curr) && isspecial(prev)) {
      match_bonus_s[j] = SPECIAL_BONUS_C[(unsigned char)prev];
    } else if (isupper(curr) && isspecial(prev)) {
      match_bonus_s[j] = SPECIAL_BONUS_C[(unsigned char)prev];
    } else if (isupper(curr) && islower(prev)) {
      match_bonus_s[j] = UPPERCASE_BONUS;
    } else if (isdigit(curr) && isspecial(prev)) {
      match_bonus_s[j] = SPECIAL_BONUS_C[(unsigned char)prev];
    } else {
      match_bonus_s[j] = 0;
    }
  }

  __syncthreads();

  size_t prev2 = 0, prev = 1, curr = 2;
  for (size_t wave = 0; wave <= (source.len - 1 + n_pattern - 1); ++wave) {
    int32_t i = wave - j;

    // swap offset before calculating
    curr = (curr + 1) % 3;
    prev = (prev + 1) % 3;
    prev2 = (prev2 + 1) % 3;

    if (i >= 0 && i < n_pattern) {
      score_t gap_penalty =
          i == n_pattern - 1 ? GAP_PENALTY_TRAILING : GAP_PENALTY_INNER;

      if (tolower(source.data[j]) == tolower(pattern[i])) {
        score_t score = SCORE_MIN;
        if (i == 0) {
          score = j * GAP_PENALTY_LEADING + match_bonus_s[j];
        } else if (j > 0) {
          score = M_s[prev2][j - 1] + ((D_s[prev2][j - 1] != SCORE_MIN)
                                           ? CONSECUTIVE_BONUS
                                           : match_bonus_s[j]);
        }

        D_s[curr][j] = score;
        M_s[curr][j] = fmax(
            D_s[curr][j], (j > 0 ? M_s[prev][j - 1] : SCORE_MIN) + gap_penalty);
      } else {
        D_s[curr][j] = SCORE_MIN;
        M_s[curr][j] = (j > 0 ? M_s[prev][j - 1] : SCORE_MIN) + gap_penalty;
      }
    }
    __syncthreads();
  }

  if (j == 0) {
    res_scores[blockIdx.x] = M_s[curr][source.len - 1];
  }
}

strings_t match(strings_t *sources, string_t pattern) {
  // TODO: figure out a better way
  if (!init) {
    hipMemcpyToSymbol(HIP_SYMBOL(SPECIAL_BONUS_C), SPECIAL_BONUS, sizeof(SPECIAL_BONUS));
    init = true;
  }

  strings_t matches = {0};

  size_t buf_size = 0;
  for (size_t i = 0; i < sources->count; ++i) {
    buf_size += sources->items[i].len;
  }

  char *buf_h, *buf_h_itr; // pinned memory
  size_t *indices_h;       // pinned memory
  bool *has_match_h;       // pinned memory
  char *buf_d, *pattern_d;
  size_t *indices_d;
  bool *has_match_d;
  size_t prev_sum, n_threads;

  hipHostMalloc(&buf_h, buf_size * sizeof(char));
  hipHostMalloc(&indices_h, sources->count * sizeof(size_t));
  hipHostMalloc(&has_match_h, sources->count * sizeof(bool));

  hipMalloc(&buf_d, buf_size * sizeof(char));
  hipMalloc(&indices_d, sources->count * sizeof(size_t));
  hipMalloc(&pattern_d, pattern.len * sizeof(char));
  hipMalloc(&has_match_d, sources->count * sizeof(bool));

  buf_h_itr = buf_h;
  prev_sum = 0, n_threads = 0;
  for (size_t i = 0; i < sources->count; ++i) {
    memcpy(buf_h_itr, sources->items[i].data,
           sources->items[i].len * sizeof(char));
    buf_h_itr += sources->items[i].len;

    indices_h[i] = prev_sum;
    prev_sum += sources->items[i].len;

    n_threads =
        sources->items[i].len > n_threads ? sources->items[i].len : n_threads;
  }
  n_threads = 1024 < n_threads ? 1024 : n_threads;

  hipMemcpy(buf_d, buf_h, buf_size * sizeof(char), hipMemcpyHostToDevice);
  hipMemcpy(indices_d, indices_h, sources->count * sizeof(size_t),
             hipMemcpyHostToDevice);
  hipMemcpy(pattern_d, pattern.data, pattern.len * sizeof(char),
             hipMemcpyHostToDevice);

  dim3 numThreads(n_threads);
  dim3 numBlocks(sources->count);
  match_kernel<<<numBlocks, numThreads>>>(buf_d, indices_d, buf_size,
                                          sources->count, pattern_d,
                                          pattern.len, has_match_d);

  hipMemcpy(has_match_h, has_match_d, sources->count * sizeof(bool),
             hipMemcpyDeviceToHost);

  for (size_t i = 0; i < sources->count; ++i) {
    if (has_match_h[i])
      da_append(matches, sources->items[i], string_t);
  }

  hipFree(buf_h);
  hipFree(indices_h);
  hipFree(has_match_h);

  hipFree(buf_d);
  hipFree(indices_d);
  hipFree(pattern_d);
  hipFree(has_match_d);

  return matches;
}

results_t score_matches(strings_t *__restrict__ matches, string_t pattern) {
  results_t res = {0};

  size_t buf_size = 0;
  for (size_t i = 0; i < matches->count; ++i) {
    buf_size += matches->items[i].len;
  }

  char *buf_h, *buf_h_itr; // pinned memory
  size_t *indices_h;       // pinned memory
  score_t *res_scores_h;   // pinned memory
  char *buf_d, *pattern_d;
  size_t *indices_d;
  score_t *res_scores_d;
  size_t prev_sum, n_threads;

  hipHostMalloc(&buf_h, buf_size * sizeof(char));
  hipHostMalloc(&indices_h, matches->count * sizeof(size_t));
  hipHostMalloc(&res_scores_h, matches->count * sizeof(score_t));

  hipMalloc(&buf_d, buf_size * sizeof(char));
  hipMalloc(&indices_d, matches->count * sizeof(size_t));
  hipMalloc(&pattern_d, pattern.len * sizeof(char));
  hipMalloc(&res_scores_d, matches->count * sizeof(score_t));

  buf_h_itr = buf_h;
  prev_sum = 0, n_threads = 0;
  for (size_t i = 0; i < matches->count; ++i) {
    memcpy(buf_h_itr, matches->items[i].data,
           matches->items[i].len * sizeof(char));
    buf_h_itr += matches->items[i].len;

    indices_h[i] = prev_sum;
    prev_sum += matches->items[i].len;

    n_threads =
        matches->items[i].len > n_threads ? matches->items[i].len : n_threads;
  }
  n_threads = 1024 < n_threads ? 1024 : n_threads;

  hipMemcpy(buf_d, buf_h, buf_size * sizeof(char), hipMemcpyHostToDevice);
  hipMemcpy(indices_d, indices_h, matches->count * sizeof(size_t),
             hipMemcpyHostToDevice);
  hipMemcpy(pattern_d, pattern.data, pattern.len * sizeof(char),
             hipMemcpyHostToDevice);

  dim3 numThreads(n_threads);
  dim3 numBlocks(matches->count);
  fused_score_kernel<<<numBlocks, numThreads>>>(buf_d, indices_d, buf_size,
                                                matches->count, pattern_d,
                                                pattern.len, res_scores_d);

  hipMemcpy(res_scores_h, res_scores_d, matches->count * sizeof(score_t),
             hipMemcpyDeviceToHost);

  for (size_t i = 0; i < matches->count; ++i) {
    scored_entry_t s = {.str = matches->items[i], .score = res_scores_h[i]};
    da_append(res, s, scored_entry_t);
  }

  hipFree(buf_h);
  hipFree(indices_h);
  hipFree(res_scores_h);

  hipFree(buf_d);
  hipFree(indices_d);
  hipFree(pattern_d);
  hipFree(res_scores_d);

  return res;
}
